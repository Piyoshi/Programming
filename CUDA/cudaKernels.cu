#include "hip/hip_runtime.h"

__global__ void func_a<<<...>>>(){} // ○

__global__ int func_b<<<...>>>(){} // ×

__global__ void func_c<<<...>>>(){ func_c<<<....>>>(); } // ×

__global__ void func_d<<<...>>>(int val, ...){} // ×


__device__ void func_da(){} // ○

__device__ int func_db(){} // ○

__device__ void func_dc(){ func_dc(); } // ×

__device__ void func_dd(int val, ...){} // ×


__host__ void func_ha(){} // ○

int func_hb(){} // ○

__host__ __device__ void func_hc(){} // ○

__host__ __global__ void func_hd(){} // ×
