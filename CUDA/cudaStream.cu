#include "hip/hip_runtime.h"

__global__ krnlFuncA(int* inA, int* inB)
{
}

__global__ krnlFuncB(int* inA, int* inB)
{
}

int main()
{
	hipStream_t streamA;
	hipStream_t streamB;

	hipStreamCreate(&streamA);
	hipStreamCreate(&streamB);

	krnlFuncA<<<..., streamA>>>(...);
	krnlFuncB<<<..., streamB>>>(...);

	hipDeviceSynchronize();
	hipStreamDestroy(&streamA);
	hipStreamDestroy(&streamB);
}
