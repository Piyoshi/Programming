#include "hip/hip_runtime.h"
#define N 64

__global__ void krnlVecAdd(int* inA, int* inB, int* inC)
{
	int index = threadIdx.x;
	inC[index] = inA[index] + inB[index];
	__syncthreads();
}


int main()
{
	int A[N], B[N], C[N];
	dim3 gd(8, 8);
	dim3 bd(16, 16, 16);
	dim3 dim3A(8, 16, 32);
	dim3 dim3B = dim3(8, 16, 32);
	dim3 dim3C = make_dim3(8, 16, 32);
	dim3 dim3D(8, 16); // dim3D(8, 16, 1);と同じ
	dim3 dim3E(8); // dim3E(8, 1, 1);と同じ
	krnlVecAdd<<<1, N>>>(A, B, C);
	//krnlVecAdd<<gd, bd>>(A, B, C);
}
