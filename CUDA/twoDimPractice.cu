#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define NX  64
#define NY  64

__global__ void access2d(float *A, float *B, float *C);

int main(void)
{
    int i, j, k;
    float A[NY][NX];
    float B[NY][NX];

    for (j = 0; j < NY; j++) {
        for (i = 0; i < NX; i++) {
            A[j][i] = 1.0;
            k = NX * j + i;
            B[k] = 1.0;
        }
    }
}


__global__ void access2d(float *A, float *B, float *C)
{
    int i, jx, jy;
    jx = blockDim.x * blockIdx.x + threadIdx.x;
    jy = blockDim.y * blockIdx.y + threadIdx.y;
    i = NX * jy + jx;
    C[i] = A[i] + B[i];
}
